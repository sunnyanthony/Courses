#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <omp.h>
#include <time.h>
#include <sys/time.h>

const int INF = 10000000;
const int V = 10010;
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B,int numdevs);
int ceil(int a, int b);
void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);
int init_device();

int n, m;	// Number of vertices, edges
static int Dist[V][V];
int *gpudist0;
int *gpudist1;
int *hostdist;
int *hostdist2;

//time
double comtime[2];
double cmitime=0;
double ccomtime[2];
double memcpy_t=0;


__global__ void calculat(int B, int n, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int *gpudist){
	//for (int b_i =  block_start_x; b_i < block_end_x; ++b_i) {
	//	for (int b_j = block_start_y; b_j < block_end_y; ++b_j) {
			// To calculate B*B elements in the block (b_i, b_j)
			// For each block, it need to compute B times
	//		for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
				// To calculate original index of elements in the block (b_i, b_j)
				// For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
	int b_i = blockIdx.x+block_start_x;
	int b_j = blockIdx.y+block_start_y;
	int distij,distik,distkj;// register value
	int block_internal_start_x = b_i * B;
	int block_internal_start_y = b_j * B; 
	int j = block_internal_start_y + threadIdx.x; // set column
	if(j > n-1) j=n-1;
	for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
				distkj = gpudist[k*n+j];	
		for (int blk_idx = 0;blk_idx<B;blk_idx++){
				//int block_internal_start_x = b_i * B;
	//			int block_internal_end_x   = (b_i +1) * B;
				//int block_internal_start_y = b_j * B; 
	//			int block_internal_end_y   = (b_j +1) * B;

	//			if (block_internal_end_x > n)	block_internal_end_x = n;
	//			if (block_internal_end_y > n)	block_internal_end_y = n;

	//			for (int i = block_internal_start_x; i < block_internal_end_x; ++i) {
	//				for (int j = block_internal_start_y; j < block_internal_end_y; ++j) {
				int i = block_internal_start_x + blk_idx;
				//int j = block_internal_start_y + threadIdx.x; // set column
				if(i > n-1) i=n-1;
				
				
				distij = gpudist[i*n+j];
				distik = gpudist[i*n+k];
					if (distik + distkj < distij)
						gpudist[i*n+j] = distik + distkj;
				__syncthreads();
		}
	}
	
			//}
		//}
	//}
}
static __global__ void calculat32(int B, int n, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int *dist){

	int b_i = blockIdx.x+block_start_x;
        int b_j = blockIdx.y+block_start_y;
        int distij,distik,distkj;// register value
        int block_internal_start_x = b_i * B;
        int block_internal_start_y = b_j * B;
        int j = block_internal_start_y + threadIdx.y;
        int i = block_internal_start_x + threadIdx.x;
        if(i > n-1) i=n-1;
        if(j > n-1) j=n-1;
            distij = dist[i*n+j];
                for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
                        distik = dist[i*n+k];
                        distkj = dist[k*n+j];
                        if (distik + distkj < distij){
                                distij = distik + distkj;
                                dist[i*n+j]=distij;
                        }
                        __syncthreads();
                }

}


double timer(void)
{	struct timeval tv;
	struct timezone tz;
  
  	double t;

  	gettimeofday(&tv, &tz);

  	t = (double)tv.tv_sec*1000;
  	t += ((double)tv.tv_usec)/1000.0;

  	return t/1000;
}


int main(int argc, char* argv[])
{
	hipError_t err;
	
	
	//time
	double t_st,t_end;
	struct timeval tv;
	struct timezone tz;
	clock_t t0,t1;
	t_st = timer();
	t0 = clock();
	double memcpy_start_t = 0, memcpy_end_t = 0;
	double IO_start_t = 0, IO_end_t = 0;
	double IO_t=0;
	//time
	
	
	int numdevs; // get devices number
	//printf("go~%d\n",argc);
	IO_start_t = timer();
	input(argv[1]);
	IO_end_t = timer();
	IO_t = IO_t + IO_end_t -IO_start_t;
	int B = atoi(argv[3]);
	//printf("%d",B);
	
	
	//init gpu && openMP threads
	numdevs = init_device();
	omp_set_num_threads(numdevs);
	
	//gpudist = (int *)malloc(sizeof(int)*2);
	//allocate GPU memory
	memcpy_start_t = timer();
	err = hipMalloc((void**)&gpudist0,sizeof(int)*n*n);
	//printf("cudaMallocgpudist0 %s\n",err);
	if(numdevs>1){
	hipSetDevice(1);
	err = hipMalloc((void**)&gpudist1,sizeof(int)*n*n);
	//printf("cudaMallocgpudist1 %s\n",err);
	}
	//copy DIST to Device(GPU)
	hipSetDevice(0);
	hipMemcpy(gpudist0,hostdist,sizeof(int)*n*n,hipMemcpyHostToDevice);
	// copy to master GPU and copy form master to slave
	memcpy_end_t = timer();
	memcpy_t = memcpy_t +memcpy_end_t - memcpy_start_t;
	
	
	
	block_FW(B,numdevs);
	
	
	memcpy_start_t = timer();
	hipMemcpy(hostdist,gpudist0,sizeof(int)*n*n,hipMemcpyDeviceToHost);
	memcpy_end_t = timer();
	memcpy_t = memcpy_t +memcpy_end_t - memcpy_start_t;
	
	
	hipFree(gpudist0);
	hipFree(gpudist1);
	IO_start_t = timer();
	output(argv[2]);
	IO_end_t = timer();
	IO_t = IO_t + IO_end_t -IO_start_t;
	
	//free(gpudist);
	free(hostdist);
	free(hostdist2);

	t_end = timer();
	t1 = clock();
	//printf("blocksize:%d Total time = %lf sec   %f \n", B, (t_end - t_st),ccomtime[0]);
	//printf("computation time0 = %f  computation time1 = %f   IO = %f  communication = %f  memcpy = %f\n", comtime[0],comtime[1], IO_t, cmitime, memcpy_t);
	
	return 0;
}

void input(char *inFileName)
{
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);
	//allocate nxn matrix
	hostdist = (int *)malloc(sizeof(int)*n*n);
	hostdist2 = (int *)malloc(sizeof(int)*n*n);
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (i == j){	
					hostdist2[i*n+j] = 0;
					hostdist[i*n+j]=0;
			}
			else{		
					hostdist2[i*n+j] = INF;
					hostdist[i*n+j]=INF;
			}
		}
	}

	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		--a, --b;
		hostdist2[a*n+b] = v;
		hostdist[a*n+b] = v;
	}
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (hostdist[i*n+j] >= INF)	fprintf(outfile, "INF ");
			else					fprintf(outfile, "%d ", hostdist[i*n+j]);
		}
		fprintf(outfile, "\n");
	}		
}

int ceil(int a, int b)
{
	return (a + b -1)/b;
}

void block_FW(int B, int numdevs)
{	omp_set_num_threads(numdevs);
	comtime[0]=0;
	comtime[1]=0;
	/*cudaEvent_t start, stop;
	cudaEventCreate(&start);
	cudaEventCreate(&stop);*/
	float time;
	double str;
	int round = ceil(n, B);//?�n/B?��???	
	//printf("round %d\n",round);
	for (int r = 0; r < round; ++r) {
		/*Phase 1*/
		//cudaEventRecord(start);
		str=timer();
		cal(B,	r,	r,	r,	1,	1);
	//	printf("phase1\n");
		/*Phase 2*/
		cal(B, r,     r,     0,             r,             1);
		cal(B, r,     r,  r +1,  round - r -1,             1);
		cal(B, r,     0,     r,             1,             r);
		cal(B, r,  r +1,     r,             1,  round - r -1);
	//	printf("phase2\n");
		
		/*cudaDeviceSynchronize();
		cudaEventRecord(stop);
		cudaEventSynchronize(stop);
		cudaEventElapsedTime(&time, start, stop);*/
		
		comtime[1] +=(timer()-str);
		comtime[0] +=(timer()-str);
		
		
		str = timer();
		if(numdevs >1 ){
			hipMemcpy(gpudist1,gpudist0,sizeof(int)*n*n,hipMemcpyDeviceToDevice);
	//		printf("D2D\n");
		}
		//cudaMemcpy(hostdist,gpudist0,sizeof(int)*n*n,cudaMemcpyDeviceToHost);
		cmitime = cmitime +timer() - str;
		
		
		/*Phase 3*/
		#pragma omp parallel private(str) 
		{
			//cudaEventRecord(start);
			str=timer();
			if(omp_get_num_threads()==1){
			cal(B, r,     0,     0,            r,             r);
			cal(B, r,     0,  r +1,  round -r -1,             r);
			cal(B, r,  r +1,     0,            r,  round - r -1);
			cal(B, r,  r +1,  r +1,  round -r -1,  round - r -1);
			}
			else{
				
				if(omp_get_thread_num()==0){
				hipSetDevice(omp_get_thread_num());
				cal(B, r,     0,     0,            r,             r);
				cal(B, r,     0,  r +1,  round -r -1,             r);
				}
				if(omp_get_thread_num()==1){
				hipSetDevice(omp_get_thread_num());
				cal(B, r,  r +1,     0,            r,  round - r -1);
				cal(B, r,  r +1,  r +1,  round -r -1,  round - r -1);
				}
				
			}
				hipDeviceSynchronize();
				comtime[omp_get_thread_num()] +=(timer()-str);
				/*cudaEventRecord(stop);
				cudaEventSynchronize(stop);
				cudaEventElapsedTime(&time, start, stop);
				comtime[omp_get_thread_num()] +=time;*/
				
		}
			
		if(numdevs >1 ){
		
			str = timer();
			hipSetDevice(1);
			hipMemcpy(hostdist2,gpudist1,sizeof(int)*n*n,hipMemcpyDeviceToHost);
			hipSetDevice(0);
			hipMemcpy(hostdist,gpudist0,sizeof(int)*n*n,hipMemcpyDeviceToHost);
			cmitime = cmitime +timer() - str;
			int j=0;
			int i=0;
			ccomtime[0]=0;
			ccomtime[1]=0;
			str=timer();
			#pragma omp parallel private(j,str) 
			{
				#pragma omp for schedule(dynamic)
				for(i=0;i<n;i++)
					for(j=0;j<n;j++){
						if(hostdist[i*n+j]>hostdist2[i*n+j]){
							hostdist[i*n+j]=hostdist2[i*n+j];
							//printf("1dist[%d][%d]= %d  ",i,j,hostdist[i*n+j]);
							//printf("2dist[%d][%d]= %d  ",i,j,hostdist2[i*n+j]);
						}
						//if(hostdist[i*n+j]==hostdist2[i*n+j])
							//printf("fuck\n");
						
						
					}
			}
			ccomtime[0] +=(timer()-str);
			//printf("  comtime[1] %f  ",comtime[1]);
			str = timer();
			hipMemcpy(gpudist0,hostdist,sizeof(int)*n*n,hipMemcpyHostToDevice);
			memcpy_t = memcpy_t +timer() - str;
		}
	}
}

void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height)
{
	//int block_end_x = block_start_x + block_height;
	//int block_end_y = block_start_y + block_width;
	dim3 guid_size = dim3(block_height, block_width);
	//int block_size = B;
	dim3 block_size;
	if(B<=32)
		block_size = dim3(B,B);
	else
		block_size = dim3(B,1);
	//for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
		//printf("a=%d",k);
		//printf("is = %d",omp_get_thread_num());
		if(omp_get_thread_num()){
			if(B<=32)
				calculat32<<<guid_size,block_size>>>(B,n,Round,block_start_x,block_start_y,block_width,block_height,gpudist1);
			else
				calculat<<<guid_size,block_size>>>(B,n,Round,block_start_x,block_start_y,block_width,block_height,gpudist1);
			//printf("no");
		}else{
			if(B<=32)
				calculat32<<<guid_size,block_size>>>(B,n,Round,block_start_x,block_start_y,block_width,block_height,gpudist0);
			else
				calculat<<<guid_size,block_size>>>(B,n,Round,block_start_x,block_start_y,block_width,block_height,gpudist0);
			//printf("yes");
		}
	//}


}

int init_device(){
	hipError_t err;
	int numdevs;
	//printf("go");
	hipGetDeviceCount(&numdevs);
	if(numdevs > 0){
	err = hipSetDevice(0);
	//printf("suda set =%s\n",err);
	}
	//printf("numdevs=%d\n",numdevs);
	
	return numdevs;
}

