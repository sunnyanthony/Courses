#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <time.h>
#include <sys/time.h>

const int INF = 10000000;
const int V = 10010;
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B);
int ceil(int a, int b);
void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);
void init_device();

int n, m;	// Number of vertices, edges
static int Dist[V][V];
int *gpudist;
int *hostdist;

//time
//double comtime=0;
//double cmitime=0;

//cudaError_t cudaEventElapsedTime(float* ms,cudaEvent_t start,cudaEvent_t end)

__global__ void calculat(int B, int n, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int *gpudist){
	//for (int b_i =  block_start_x; b_i < block_end_x; ++b_i) {
	//	for (int b_j = block_start_y; b_j < block_end_y; ++b_j) {
			// To calculate B*B elements in the block (b_i, b_j)
			// For each block, it need to compute B times
	//		for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
				// To calculate original index of elements in the block (b_i, b_j)
				// For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
	int b_i = blockIdx.x+block_start_x;
	int b_j = blockIdx.y+block_start_y;
	int distij,distik,distkj;// register value
	for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
		for (int blk_idx = 0;blk_idx<B;blk_idx++){
				int block_internal_start_x = b_i * B;
	//			int block_internal_end_x   = (b_i +1) * B;
				int block_internal_start_y = b_j * B; 
	//			int block_internal_end_y   = (b_j +1) * B;

	//			if (block_internal_end_x > n)	block_internal_end_x = n;
	//			if (block_internal_end_y > n)	block_internal_end_y = n;

	//			for (int i = block_internal_start_x; i < block_internal_end_x; ++i) {
	//				for (int j = block_internal_start_y; j < block_internal_end_y; ++j) {
				int i = block_internal_start_x + blk_idx;
				int j = block_internal_start_y + threadIdx.x; // set column
				if(i > n-1) i=n-1;
				if(j > n-1) j=n-1;
				
				distij = gpudist[i*n+j];
				distik = gpudist[i*n+k];
				distkj = gpudist[k*n+j];	
					if (distik + distkj < distij)
						gpudist[i*n+j] = distik + distkj;
				__syncthreads();
		}
	}
	
			//}
		//}
	//}
}
static __global__ void calculat32(int B, int n, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int *dist){

		int b_i = blockIdx.x+block_start_x;
        int b_j = blockIdx.y+block_start_y;
        int distij,distik,distkj;// register value
        int block_internal_start_x = b_i * B;
        int block_internal_start_y = b_j * B;
        int j = block_internal_start_y + threadIdx.y;
        int i = block_internal_start_x + threadIdx.x;
        if(i > n-1) i=n-1;
        if(j > n-1) j=n-1;
            distij = dist[i*n+j];
                for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
                        distik = dist[i*n+k];
                        distkj = dist[k*n+j];
                        if (distik + distkj < distij){
                                distij = distik + distkj;
                                dist[i*n+j]=distij;
                        }
                        __syncthreads();
                }

}

double timer(void)
{	struct timeval tv;
	struct timezone tz;
  
  	double t;

  	gettimeofday(&tv, &tz);

  	t = (double)tv.tv_sec*1000;
  	t += ((double)tv.tv_usec)/1000.0;

  	return t/1000;
}

int main(int argc, char* argv[])
{	hipError_t err;
	
	//time
	/*double t_st,t_end;
	struct timeval tv;
	struct timezone tz;
	clock_t t0,t1;
	t_st = timer();
	t0 = clock();
	double memcpy_start_t = 0, memcpy_end_t = 0;
	double memcpy_t=0;
	double IO_start_t = 0, IO_end_t = 0;
	double IO_t=0;
	
	//IO
	IO_start_t = timer();*/
	input(argv[1]);
	//IO_end_t = timer();
	//IO_t = IO_t + IO_end_t -IO_start_t;
	
	int B = atoi(argv[3]);
	//printf("%d",B);
	
	
	//init gpu
	init_device();
	//printf("init_device %d \n",B);
	
	
	//allocate GPU memory
	//memcpy_start_t = timer();
	hipMalloc((void**)&gpudist,sizeof(int)*n*n);
	//printf("cudaMalloc %s \n",err);
	//copy DIST to Device(GPU)
	hipMemcpy(gpudist,hostdist,sizeof(int)*n*n,hipMemcpyHostToDevice);
	//memcpy_end_t = timer();
	//memcpy_t = memcpy_t +memcpy_end_t - memcpy_start_t;
	//printf("cudaMemcpy %d \n",B);
	
	
	block_FW(B);
	
	
	//memcpy_start_t = timer();
	hipMemcpy(hostdist,gpudist,sizeof(int)*n*n,hipMemcpyDeviceToHost);
	//memcpy_end_t = timer();
	//memcpy_t = memcpy_t +memcpy_end_t - memcpy_start_t;
	
	
	
	hipFree(gpudist);
	
	//IO
	//IO_start_t = timer();
	output(argv[2]);
	//IO_end_t = timer();
	//IO_t = IO_t + IO_end_t -IO_start_t;
	
	
	free(hostdist);
	
	//gettimeofday(&tv, &tz);
	//t_end = timer();
	//t1 = clock();
	//printf("#b:%d Total Cuda time = %lf sec (%lf)\n" , B, (t_end - t_st),(t1-t0)/(double)(CLOCKS_PER_SEC));
	//printf("computation time = %f  IO = %f  communication = %f  memcpy = %f\n", comtime/1000, IO_t, cmitime, memcpy_t);
	
	return 0;
}

void input(char *inFileName)
{
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);
	//allocate nxn matrix
	hostdist = (int *)malloc(sizeof(int)*n*n);
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (i == j){	
					hostdist[i*n+j]=0;}
			else{		
					hostdist[i*n+j]=INF;}
		}
	}

	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		--a, --b;
		
		hostdist[a*n+b] = v;
	}
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (hostdist[i*n+j] >= INF)	fprintf(outfile, "INF ");
			else					fprintf(outfile, "%d ", hostdist[i*n+j]);
		}
		fprintf(outfile, "\n");
	}		
}

int ceil(int a, int b)
{
	return (a + b -1)/b;
}

void block_FW(int B)
{	
	//cudaEvent_t start, stop;
	//cudaEventCreate(&start);
	//cudaEventCreate(&stop);
	//float time;
	
	
	int round = ceil(n, B);//?�n/B?��???	
	for (int r = 0; r < round; ++r) {
		/* Phase 1*/
		//cudaEventRecord(start);
		cal(B,	r,	r,	r,	1,	1);
		//cudaEventRecord(stop);
		//cudaEventSynchronize(stop);
		//cudaEventElapsedTime(&time, start, stop);
		//comtime +=time;
		
		/* Phase 2*/
		//cudaEventRecord(start);
		cal(B, r,     r,     0,             r,             1);
		cal(B, r,     r,  r +1,  round - r -1,             1);
		cal(B, r,     0,     r,             1,             r);
		cal(B, r,  r +1,     r,             1,  round - r -1);
		//cudaDeviceSynchronize();
		//cudaEventRecord(stop);
		//cudaEventSynchronize(stop);
		//cudaEventElapsedTime(&time, start, stop);
		//comtime +=time;
		hipDeviceSynchronize();
		/* Phase 3*/
		//cudaEventRecord(start);
		cal(B, r,     0,     0,            r,             r);
		cal(B, r,     0,  r +1,  round -r -1,             r);
		cal(B, r,  r +1,     0,            r,  round - r -1);
		cal(B, r,  r +1,  r +1,  round -r -1,  round - r -1);
		hipDeviceSynchronize();
		//cudaDeviceSynchronize();
		//cudaDeviceSynchronize();
		//cudaEventRecord(stop);
		//cudaEventSynchronize(stop);
		//cudaEventElapsedTime(&time, start, stop);
		//comtime +=time;
	}
}

void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height)
{
	//int block_end_x = block_start_x + block_height;
	//int block_end_y = block_start_y + block_width;
	dim3 guid_size = dim3(block_height, block_width);
	dim3 block_size;
	if(B<=32)
		block_size = dim3(B,B);
	else
		block_size = dim3(B,1);
		//printf("a=%d\n",k);
	if(B<=32)	
		calculat32<<<guid_size,block_size>>>(B,n,Round,block_start_x,block_start_y,block_width,block_height,gpudist);
	else
		calculat<<<guid_size,block_size>>>(B,n,Round,block_start_x,block_start_y,block_width,block_height,gpudist);
}

void init_device(){
	int numdevs;
	//printf("go");
	hipGetDeviceCount(&numdevs);
	if(numdevs > 0){
	hipSetDevice(0);
	}
	//printf("numdevs=%d\n",numdevs);
}


