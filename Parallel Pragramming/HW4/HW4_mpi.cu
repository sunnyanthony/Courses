#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hip/hip_runtime_api.h"
#include <mpi.h>
#include <omp.h>
#include <time.h>
#include <sys/time.h>

#define result_tag 11

const int INF = 10000000;
const int V = 10010;
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B,int numdevs);
int ceil(int a, int b);
void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height);
int init_device();

int n, m;	// Number of vertices, edges
static int Dist[V][V];
int *gpudist0;
int *gpudist1;
int *hostdist;
int *hostdist2;


//MPI
int size, rank;
MPI_Status status;
MPI_Request status_req[2];

//time
/*double comtime=0;
double cmitime=0;
double memcpy_t=0;*/



__global__ void calculat(int B, int n, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int *gpudist){
	//for (int b_i =  block_start_x; b_i < block_end_x; ++b_i) {
	//	for (int b_j = block_start_y; b_j < block_end_y; ++b_j) {
			// To calculate B*B elements in the block (b_i, b_j)
			// For each block, it need to compute B times
	//		for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
				// To calculate original index of elements in the block (b_i, b_j)
				// For instance, original index of (0,0) in block (1,2) is (2,5) for V=6,B=2
	int b_i = blockIdx.x+block_start_x;
	int b_j = blockIdx.y+block_start_y;
	int distij,distik,distkj;// register value
	int block_internal_start_x = b_i * B;
	int block_internal_start_y = b_j * B; 
	int j = block_internal_start_y + threadIdx.x; // set column
	if(j > n-1) j=n-1;
	for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
		distkj = gpudist[k*n+j];	
		for (int blk_idx = 0;blk_idx<B;blk_idx++){
				//int block_internal_start_x = b_i * B;
	//			int block_internal_end_x   = (b_i +1) * B;
				//int block_internal_start_y = b_j * B; 
	//			int block_internal_end_y   = (b_j +1) * B;

	//			if (block_internal_end_x > n)	block_internal_end_x = n;
	//			if (block_internal_end_y > n)	block_internal_end_y = n;

	//			for (int i = block_internal_start_x; i < block_internal_end_x; ++i) {
	//				for (int j = block_internal_start_y; j < block_internal_end_y; ++j) {
				int i = block_internal_start_x + blk_idx;
				//int j = block_internal_start_y + threadIdx.x; // set column
				if(i > n-1) i=n-1;
				
				
				distij = gpudist[i*n+j];
				distik = gpudist[i*n+k];
					if (distik + distkj < distij)
						gpudist[i*n+j] = distik + distkj;
				__syncthreads();
		}
	}
	
			//}
		//}
	//}
}
static __global__ void calculat32(int B, int n, int Round, int block_start_x, int block_start_y, int block_width, int block_height, int *dist){

		int b_i = blockIdx.x+block_start_x;
        int b_j = blockIdx.y+block_start_y;
        int distij,distik,distkj;// register value
        int block_internal_start_x = b_i * B;
        int block_internal_start_y = b_j * B;
        int j = block_internal_start_y + threadIdx.y;
        int i = block_internal_start_x + threadIdx.x;
        if(i > n-1) i=n-1;
        if(j > n-1) j=n-1;
            distij = dist[i*n+j];
                for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
                        distik = dist[i*n+k];
                        distkj = dist[k*n+j];
                        if (distik + distkj < distij){
                                distij = distik + distkj;
                                dist[i*n+j]=distij;
                        }
                        __syncthreads();
                }

}


double timer(void)
{	struct timeval tv;
	struct timezone tz;
  
  	double t;

  	gettimeofday(&tv, &tz);

  	t = (double)tv.tv_sec*1000;
  	t += ((double)tv.tv_usec)/1000.0;

  	return t/1000;
}

int main(int argc, char* argv[])
{
	//MPI init
	
	MPI_Init(&argc,&argv);
	MPI_Comm_size (MPI_COMM_WORLD, &size); 
	MPI_Comm_rank (MPI_COMM_WORLD, &rank); 
	MPI_Barrier(MPI_COMM_WORLD);
	
	/*
	//time
	double t_st,t_end;
	struct timeval tv;
	struct timezone tz;
	clock_t t0,t1;
	t_st = timer();
	t0 = clock();
	double memcpy_start_t = 0, memcpy_end_t = 0;
	double IO_start_t = 0, IO_end_t = 0;
	double IO_t=0;
	//time
	*/
	
	hipError_t err;
	int numdevs; // get devices number
	//printf("go~%d\n",argc);
	//IO_start_t = timer();
	input(argv[1]);
	/*IO_end_t = timer();
	IO_t = IO_t + IO_end_t -IO_start_t;
	*/
	int B = atoi(argv[3]);
	//printf("%d",B);
	
	
	//init gpu && openMP threads
	numdevs = init_device();
	//omp_set_num_threads(numdevs);
	
	//gpudist = (int *)malloc(sizeof(int)*2);
	//allocate GPU memory
	//err = hipMalloc((void**)&gpudist0,sizeof(int)*n*n);
	//printf("cudaMallocgpudist0 %s\n",err);
	//if(size>1 && rank ==1){
	hipSetDevice(rank);
	//memcpy_start_t = timer();
	hipMalloc((void**)&gpudist0,sizeof(int)*n*n);
	//printf("cudaMallocgpudist1 %s\n",err);
	//}
	//copy DIST to Device(GPU)
	//hipSetDevice(0);
	hipMemcpy(gpudist0,hostdist,sizeof(int)*n*n,hipMemcpyHostToDevice);
	/*memcpy_end_t = timer();
	memcpy_t = memcpy_t +memcpy_end_t - memcpy_start_t;*/
	
	
	block_FW(B,numdevs);
	
	
	//memcpy_start_t = timer();
	hipMemcpy(hostdist,gpudist0,sizeof(int)*n*n,hipMemcpyDeviceToHost);
	/*memcpy_end_t = timer();
	memcpy_t = memcpy_t +memcpy_end_t - memcpy_start_t;*/
	
	hipFree(gpudist0);
	//hipFree(gpudist1);
	//IO_start_t = timer();
	if(rank==0)
		output(argv[2]);
	//IO_end_t = timer();
	//IO_t = IO_t + IO_end_t -IO_start_t;
	
	//free(gpudist);
	free(hostdist);
	free(hostdist2);
	
	/*t_end = timer();
	t1 = clock();
	//printf("#b:%d Total Cuda[%d] time = %lf sec (%lf)\n" , B,rank, (t_end - t_st),(t1-t0)/(double)(CLOCKS_PER_SEC));
	//printf("Cuda[%d] computation time = %f   IO = %f  communication = %f  memcpy = %f\n",rank, comtime, IO_t, cmitime, memcpy_t);
	*/
	MPI_Barrier(MPI_COMM_WORLD);
	MPI_Finalize();
	return 0;
}

void input(char *inFileName)
{
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);
	//allocate nxn matrix
	hostdist = (int *)malloc(sizeof(int)*n*n);
	hostdist2 = (int *)malloc(sizeof(int)*n*n);
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (i == j){	
					hostdist2[i*n+j] = 0;
					hostdist[i*n+j]=0;
			}
			else{		
					hostdist2[i*n+j] = INF;
					hostdist[i*n+j]=INF;
			}
		}
	}

	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		--a, --b;
		hostdist2[a*n+b] = v;
		hostdist[a*n+b] = v;
	}
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	if(rank == 0){
		for (int i = 0; i < n; ++i) {
			for (int j = 0; j < n; ++j) {
				if (hostdist[i*n+j] >= INF)	fprintf(outfile, "INF ");
				else					fprintf(outfile, "%d ", hostdist[i*n+j]);
			}
			fprintf(outfile, "\n");
		}		
	}
}

int ceil(int a, int b)
{
	return (a + b -1)/b;
}

void block_FW(int B, int numdevs)
{
	//omp_set_num_threads(B);
	double str;



	int round = ceil(n, B);//?�n/B?��???	
	//printf("round %d\n",round);
	for (int r = 0; r < round; ++r) {
		/* Phase 1*/
		//str=timer();
		cal(B,	r,	r,	r,	1,	1);
		//printf("phase1\n");
		/* Phase 2*/
		cal(B, r,     r,     0,             r,             1);
		cal(B, r,     r,  r +1,  round - r -1,             1);
		cal(B, r,     0,     r,             1,             r);
		cal(B, r,  r +1,     r,             1,  round - r -1);
		//printf("phase2\n");
		hipDeviceSynchronize();
		//comtime +=(timer()-str);
		//if(numdevs >1 ){
		//	hipMemcpy(gpudist1,gpudist0,sizeof(int)*n*n,hipMemcpyDeviceToDevice);
		//	printf("D2D\n");
		//}
		//hipMemcpy(hostdist,gpudist0,sizeof(int)*n*n,hipMemcpyDeviceToHost);
		//output("iniout");
		/* Phase 3*/
		//#pragma omp parallel
		//{
		//str=timer();
			if(size==1){
			cal(B, r,     0,     0,            r,             r);
			cal(B, r,     0,  r +1,  round -r -1,             r);
			cal(B, r,  r +1,     0,            r,  round - r -1);
			cal(B, r,  r +1,  r +1,  round -r -1,  round - r -1);
			}
			else{
				hipSetDevice(rank);
				cal(B, r,  (r +1)*rank,     0,            r,  (round - r -1)*rank-r*(rank-1));
				cal(B, r,  (r +1)*rank,  r +1,  round -r -1,  (round - r -1)*rank-r*(rank-1));
				/*
				(round - r -1)*rank-r*(rank-1)
				if rank ==0 then (round - r -1)*0-r*(0-1) => r
				if rank ==1 then (round - r -1)*1-r*(1-1) => round - r -1
				*/
				hipDeviceSynchronize();
			}
			//comtime +=(timer()-str);
			
		//}
	//str = timer();
		if(size >1 ){
			hipSetDevice(rank);
			//hipMemcpy(hostdist2,gpudist1,sizeof(int)*n*n,hipMemcpyDeviceToHost);
			//hipSetDevice(0);
			hipMemcpy(hostdist,gpudist0,sizeof(int)*n*n,hipMemcpyDeviceToHost);
			int j=0;
			int i=0;
			memset(hostdist2,0,sizeof(int)*n*n);
			//MPI_Barrier(MPI_COMM_WORLD);
			//printf("will send");
			/*if(rank==1){
				MPI_Irecv(hostdist2,n*n,MPI_INT,1,result_tag,MPI_COMM_WORLD,&status_req[0]);
				MPI_Isend(hostdist,n*n,MPI_INT,0,result_tag,MPI_COMM_WORLD,&status_req[1]);
				printf("will recive");
			}
			if(rank==0){
				MPI_Irecv(hostdist2,n*n,MPI_INT,0,result_tag,MPI_COMM_WORLD,&status_req[0]);
				MPI_Isend(hostdist,n*n,MPI_INT,1,result_tag,MPI_COMM_WORLD,&status_req[1]);
				printf("will recive");
			}
			MPI_Waitall(1,status_req,&status);*/
			if(rank==1){
				MPI_Send(hostdist,n*n,MPI_INT,size-1-rank,result_tag,MPI_COMM_WORLD);
				MPI_Recv(hostdist2,n*n,MPI_INT,MPI_ANY_SOURCE,result_tag,MPI_COMM_WORLD,&status);
			}
			else{
				MPI_Recv(hostdist2,n*n,MPI_INT,MPI_ANY_SOURCE,result_tag,MPI_COMM_WORLD,&status);
				MPI_Send(hostdist,n*n,MPI_INT,size-1-rank,result_tag,MPI_COMM_WORLD);
			}
			
			//cmitime = cmitime +timer() - str;
			
			//			str = timer();
			#pragma omp parallel private(j) 
			{
				#pragma omp for schedule(dynamic)
				for(i=0;i<n;i++)
					for(j=0;j<n;j++){
						if(hostdist[i*n+j]>hostdist2[i*n+j]){
							hostdist[i*n+j]=hostdist2[i*n+j];
							//printf("1dist[%d][%d]= %d  ",i,j,hostdist[i*n+j]);
							//printf("2dist[%d][%d]= %d  ",i,j,hostdist2[i*n+j]);
						}
						//if(hostdist[i*n+j]==hostdist2[i*n+j])
							//printf("fuck\n");
					}
			}
			
			//comtime+=(timer()-str);
			//str = timer();
			hipMemcpy(gpudist0,hostdist,sizeof(int)*n*n,hipMemcpyHostToDevice);
			//memcpy_t = memcpy_t +timer() - str;
		}
	}
}

void cal(int B, int Round, int block_start_x, int block_start_y, int block_width, int block_height)
{
	//int block_end_x = block_start_x + block_height;
	//int block_end_y = block_start_y + block_width;
	dim3 guid_size = dim3(block_height, block_width);
	//dim3 block_size = dim3(B,B);
	dim3 block_size;
	if(B<=32)
		block_size = dim3(B,B);
	else
		block_size = dim3(B,1);
		
		if(B<=32)
				calculat32<<<guid_size,block_size>>>(B,n,Round,block_start_x,block_start_y,block_width,block_height,gpudist0);
		else	calculat<<<guid_size,block_size>>>(B,n,Round,block_start_x,block_start_y,block_width,block_height,gpudist0);

	


}

int init_device(){
	hipError_t err;
	int numdevs;
	//printf("go");
	hipGetDeviceCount(&numdevs);
	if(numdevs > 1){
	err = hipSetDevice(rank);
	//printf("suda set =%s\n",err);
	}
	else
		hipSetDevice(0);
	//printf("numdevs=%d\n",numdevs);
	
	return numdevs;
}

